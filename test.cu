
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello from GPU, thread %d\n", threadIdx.x);
}

int main() {
    printf("Hello from CPU\n");
    helloFromGPU<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}